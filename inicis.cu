#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

__global__ reverseArray(int *array, int *arraytr){
    int inOffset  = blockDim.x * blockIdx.x;
    int outOffset = blockDim.x * (gridDim.x - 1 - blockIdx.x);
    int in  = inOffset + threadIdx.x;
    int out = outOffset + (blockDim.x - 1 - threadIdx.x);
    array[out] = arraytr[in];
}


int main(int argc, char** argv){	

	int dimA = 256 * 1024; 
	int h_a[],h_b[];
	int *d_a, *d_b;

	int numThreadsPerBlock = 256;
	int numBlocks = 1024;

	hipMalloc(&d_a, dimA * sizeof(int));
	hipMalloc(&d_b, dimA * sizeof(int));

	for (int i = 0; i < dimA; ++i)
	{
		h_a[i] = i;
	}

	hipMemcpy(d_a, h_a, dimA * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid(numBlocks);
	dim3 dimBlock(numThreadsPerBlock);

	reverseArray <<<dimGrid, dimBlock>>>(d_a, d_b);
	hipDeviceSynchronize();

	CudaCheckError('Kernel Invocation');

	hipMemcpy(h_b, d_b, dimA * sizeof(int), hipMemcpyDeviceToHost);

	CudaCheckError('Copy data error');

	for (int i = 0; i < 256*1024; ++i)
	{
		printf("%d\n", h_b[i]);
	}

	hipFree(d_a);
	hipFree(d_b);

	CudaCheckError('Could not free memory');

	return 0;
}


void CudaCheckError(const char *msg){

	hipError_t err = hipGetLastError();
	if(hipSuccess != err){
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }     

}
